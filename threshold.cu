
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define DIM 1600
#define PI 3.14159265


__global__ void grayscale(unsigned char *R_input, unsigned char *G_input,
                        unsigned char *B_input, size_t i_size,
                        unsigned int *hist) {
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int offset = x + y * i_size;
  R_input[offset] = 0.2989 * R_input[offset] +  0.587 * G_input[offset] + 0.1140 * B_input[offset];
  G_input[offset] = 0.2989 * R_input[offset] +  0.587 * G_input[offset] + 0.1140 * B_input[offset];
  B_input[offset] = 0.2989 * R_input[offset] +  0.587 * G_input[offset] + 0.1140 * B_input[offset];
  atomicAdd( &(hist[R_input[offset]]), 1);
}
__global__ void binary(unsigned char *R_input, unsigned char *G_input,
                        unsigned char *B_input, size_t i_size,
                        int um) {
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int offset = x + y * i_size;
  unsigned char c;
  if (R_input[offset] > um) c = 255;
  else c = 0;
  R_input[offset] = c;
  G_input[offset] = c;
  B_input[offset] = c;
}
