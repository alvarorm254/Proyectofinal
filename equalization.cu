
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define DIM 1600
#define PI 3.14159265

__global__ void Get_Histogram(unsigned char *R_input, unsigned char *G_input,
                        unsigned char *B_input, size_t i_size,
                        unsigned int *hist_r,unsigned int *hist_g,unsigned int *hist_b) {

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int offset = x + y * i_size;

  atomicAdd( &(hist_r[R_input[offset]]), 1);
  atomicAdd( &(hist_g[G_input[offset]]), 1);
  atomicAdd( &(hist_b[B_input[offset]]), 1);
}

__global__ void Equalization_GPU(unsigned char *R_input, unsigned char *G_input,
                        unsigned char *B_input, size_t i_size,
                        unsigned char *r_dataE, unsigned char *g_dataE,
                        unsigned char *b_dataE,
                        unsigned int *hist_r,unsigned int *hist_g,unsigned int *hist_b) {

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int offset = x + y * i_size;
  r_dataE[offset] = hist_r[R_input[offset]];
  g_dataE[offset] = hist_g[G_input[offset]];
  b_dataE[offset] = hist_b[B_input[offset]];
}

