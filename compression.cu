
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define DIM 1600
#define PI 3.14159265
__global__ void Pixelado(unsigned char *R_input, unsigned char *G_input,unsigned char *B_input, size_t i_size, unsigned char *R_output, unsigned char *G_output,unsigned char *B_output)
{
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int offset = x + y * i_size;
  int offset2 = x-x%5 + (y-y%5) * i_size;
  R_output[offset] = R_input[offset2];
  G_output[offset] = G_input[offset2];
  B_output[offset] = B_input[offset2];
}

__global__ void Requant(unsigned char *R_input, unsigned char *G_input,unsigned char *B_input, size_t i_size, unsigned char *R_output, unsigned char *G_output,unsigned char *B_output)
{
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int offset = x + y * i_size;
  R_output[offset] = R_input[offset]-R_input[offset]%16;
  G_output[offset] = G_input[offset]-G_input[offset]%16;
  B_output[offset] = B_input[offset]-B_input[offset]%16;
}

