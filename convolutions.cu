
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define DIM 1600
#define PI 3.14159265

__global__ void erode(unsigned char *R_input, unsigned char *G_input,
                        unsigned char *B_input, size_t i_size,
                        unsigned char *r_dataC, unsigned char *g_dataC,
                        unsigned char *b_dataC, unsigned long col, unsigned long row,
                        unsigned int dim, int m) {
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int offset = x + y * i_size;
  int offset2, ximg, yimg;
  int c1 = 255,c2 = 255,c3 = 255;
  int end = dim/2, ini = -end;

  for (int i = ini; i <= end; i++) {
    ximg = x + i;
    for (int j = ini; j <= end; j++) {
      yimg = y + j;
      offset2 = ximg + yimg * i_size;
      if (ximg < col && yimg < row)
        if (ximg > 0 && yimg > 0)
          if(R_input[offset2]+G_input[offset2]+B_input[offset2]<c1+c2+c3)
          c1 = R_input[offset2];
          c2 = G_input[offset2];
          c3 = B_input[offset2];
    }
  }
  r_dataC[offset] = c1;
  g_dataC[offset] = c2;
  b_dataC[offset] = c3;
}

__global__ void dilate(unsigned char *R_input, unsigned char *G_input,
                        unsigned char *B_input, size_t i_size,
                        unsigned char *r_dataC, unsigned char *g_dataC,
                        unsigned char *b_dataC, unsigned long col, unsigned long row,
                        unsigned int dim, int m) {
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int offset = x + y * i_size;
  int offset2, ximg, yimg;
  int c1 = 0,c2 = 0,c3 = 0;
  int end = dim/2, ini = -end;

  for (int i = ini; i <= end; i++) {
    ximg = x + i;
    for (int j = ini; j <= end; j++) {
      yimg = y + j;
      offset2 = ximg + yimg * i_size;
      if (ximg < col && yimg < row)
        if (ximg > 0 && yimg > 0)
          if(R_input[offset2]+G_input[offset2]+B_input[offset2]>c1+c2+c3)
          c1 = R_input[offset2];
          c2 = G_input[offset2];
          c3 = B_input[offset2];
    }
  }
  r_dataC[offset] = c1;
  g_dataC[offset] = c2;
  b_dataC[offset] = c3;
}

__global__ void median_filter(unsigned char *R_input, unsigned char *G_input,
                        unsigned char *B_input, size_t i_size,
                        unsigned char *r_dataC, unsigned char *g_dataC,
                        unsigned char *b_dataC, unsigned long col, unsigned long row,
                        unsigned int dim) {
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int offset = x + y * i_size;
  int offset2, ximg, yimg;
  unsigned char temp_r = 0, temp_g = 0, temp_b = 0, temp;
  int end = dim/2, ini = -end, k = 0, n = 0, i, j;
  int hr[9];
  int hg[9];
  int hb[9];

  for (i = ini; i <= end; i++) {
    ximg = x + i;
    for (j = ini; j <= end; j++) {
      yimg = y + j;
      offset2 = ximg + yimg * i_size;
      if (ximg < col && yimg < row)
        if (ximg > 0 && yimg > 0) {
          hr[n] = R_input[offset2];
          hg[n] = G_input[offset2];
          hb[n] = B_input[offset2];
          n++;}
      k++;
    }
  }
  for (i = 0; i < n; i++)
    for (j= i + 1; j < n; j++)
      if (hr[j] < hr[i]) {
        temp = hr[j];
        hr[j] = hr[i];
        hr[i] = temp;}

  for (i = 0; i < n; i++)
    for (j= i + 1; j < n; j++)
      if (hg[j] < hg[i]) {
        temp = hg[j];
        hg[j] = hg[i];
        hg[i] = temp;}

  for (i = 0; i < n; i++)
    for (j= i + 1; j < n; j++)
      if (hb[j] < hb[i]) {
        temp = hb[j];
        hb[j] = hb[i];
        hb[i] = temp;}

  if(n%2 == 1){
    temp_r = hr[(n/2)];
    temp_g = hg[(n/2)];
    temp_b = hb[(n/2)];
  }else{
    temp_r = hr[(n/2)] + hr[(n/2) - 1];
    temp_g = hg[(n/2)] + hg[(n/2) - 1];
    temp_b = hb[(n/2)] + hb[(n/2) - 1];}

  r_dataC[offset] = temp_r;
  g_dataC[offset] = temp_g;
  b_dataC[offset] = temp_b;
}
__global__ void Operador_Convolucion(unsigned char *R_input, unsigned char *G_input,
                        unsigned char *B_input, size_t i_size,
                        unsigned char *r_dataC, unsigned char *g_dataC,
                        unsigned char *b_dataC, unsigned long col, unsigned long row,
                        float *mask, unsigned int dim) {

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int offset = x + y * i_size;
  int offset2, ximg, yimg;
  unsigned char temp_r = 0, temp_g = 0, temp_b = 0;
  int end = dim/2, ini = -end, k = 0;

  for (int i = ini; i <= end; i++) {
    ximg = x + i;
    for (int j = ini; j <= end; j++) {
      yimg = y + j;
      offset2 = ximg + yimg * i_size;
      if (ximg < col && yimg < row)
        if (ximg > 0 && yimg > 0) {
          temp_r += R_input[offset2]*mask[k];
          temp_g += G_input[offset2]*mask[k];
          temp_b += B_input[offset2]*mask[k];}
      k++;
    }
  }
  r_dataC[offset] = temp_r;
  g_dataC[offset] = temp_g;
  b_dataC[offset] = temp_b;
}

