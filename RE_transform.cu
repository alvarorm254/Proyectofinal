
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define DIM 1600
#define PI 3.14159265

__global__ void Rotate(uchar4 *ptr, unsigned char *R_input, unsigned char *G_input,
                        unsigned char *B_input, size_t i_size, float a,
                        unsigned long col, unsigned long row)
{
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int offset = x + y * blockDim.x * gridDim.x;
  x = x - (blockDim.x * gridDim.x / 2);
  y = y - (blockDim.y * gridDim.y / 2);

  unsigned char* f_r, *f_g, *f_b;

  int ximg = (x*cos(a) + y*sin(a)) + (col/2), yimg = (y*cos(a) - x*sin(a)) + (row/2);
  if (ximg < col && yimg < row) {
    f_r = (unsigned char*)((char*)R_input + yimg*i_size);
    f_g = (unsigned char*)((char*)G_input + yimg*i_size);
    f_b = (unsigned char*)((char*)B_input + yimg*i_size);
    ptr[offset].x = f_r[ximg];
    ptr[offset].y = f_g[ximg];
    ptr[offset].z = f_b[ximg];
    ptr[offset].w = 255;
  } else{
    ptr[offset].x = 0;
    ptr[offset].y = 0;
    ptr[offset].z = 0;
    ptr[offset].w = 255;
  }

}

__global__ void Scale(unsigned char *R_input, unsigned char *G_input,unsigned char *B_input,
                        unsigned char *R_output, unsigned char *G_output,unsigned char *B_output,
                        size_t i_size, size_t pitch2, float s,
                        unsigned long col, unsigned long row){
  float x = threadIdx.x + (blockIdx.x * blockDim.x);
  float y = threadIdx.y + (blockIdx.y * blockDim.y);
  int offset = x + y * pitch2;
  x = x - (DIM / 2);
  y = y - (DIM / 2);

  unsigned char* f_r, *f_g, *f_b;
  x /= s; y /= s;

  int ximg = x + (col/2), yimg = y + (row/2);
  if (ximg < (col - 1) && yimg < (row - 1)) {
    f_r = (unsigned char*)((char*)R_input + yimg*i_size);
    f_g = (unsigned char*)((char*)G_input + yimg*i_size);
    f_b = (unsigned char*)((char*)B_input + yimg*i_size);
    float cx = x - floor(x);
    float cy = y - floor(y);
    float R1 = f_r[ximg]*(1 - cx) + f_r[ximg + 1]*(cx);
    float R2 = f_r[ximg + i_size]*(1 - cx) + f_r[ximg + 1 + i_size]*(cx);
    R_output[offset] = R1*(1 - cy) + R2*(cy);

    R1 = f_g[ximg]*(1 - cx) + f_g[ximg + 1]*(cx);
    R2 = f_g[ximg + i_size]*(1 - cx) + f_g[ximg + 1 + i_size]*(cx);
    G_output[offset] = R1*(1 - cy) + R2*(cy);

    R1 = f_b[ximg]*(1 - cx) + f_b[ximg + 1]*(cx);
    R2 = f_b[ximg + i_size]*(1 - cx) + f_b[ximg + 1 + i_size]*(cx);
    B_output[offset] = R1*(1 - cy) + R2*(cy);
  }else{
    R_output[offset] = 0;
    G_output[offset] = 0;
    B_output[offset] = 0;
  }
}
