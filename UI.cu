#include "hip/hip_runtime.h"
//nvcc Final.cu -o out -lglut -lGLEW -lGL -lm -ccbin clang-3.8 -lstdc++
///////////////////////////////////////////////////
//          Call to general libraries            //
///////////////////////////////////////////////////
#include <cstdio>
#include <stdlib.h>
#include <stdint.h>
#include <assert.h>
#include <math.h>
#include <complex.h>
#include <string.h>
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <GL/glut.h>
#include <GL/freeglut_ext.h>
#include <cuda_gl_interop.h>
///////////////////////////////////////////////////
//            Call to cuda funtions              //
///////////////////////////////////////////////////
#include "FFT.cu"
#include "convolutions.cu"
#include "threshold.cu"
#include "RE_transform.cu"
#include "equalization.cu"
#include "noise.cu"
#include "backup.cu"
#include "compression.cu"


using namespace std;

///////////////////////////////////////////////////
//              Defining variables               //
///////////////////////////////////////////////////
#define WIDTH  1280
#define HEIGHT 960
#define DIM 1600
#define PI 3.14159265

static int sub_00;
static int sub_01;
static int sub_02;
static int sub_03;
static int sub_04;

bool Equalization = 0, Flag_Filt = 0, Flag_Med = 0, Flag_PPnoise = 0,Flag_Pix = 0,Flag_req=0;
bool Flag_Reset = 0, Flag_Ero = 0, Flag_Dil = 0, Flag_Gray = 0, Flag_BW = 0, Flag_Fourier = 0;
long long int sizeImage;
float Scale_Factor;
float Rotation_Factor;
unsigned long widht, height;
int Num_Cols, Num_Rows, Dim_Con, Num_Rows_Fourier, Num_Cols_Fourier, Max_E;
size_t or_size, mor_size, equ_size, fou_size;
unsigned char *Image_R, *Image_G, *Image_B;
unsigned char *Image_R_bk, *Image_G_bk, *Image_B_bk;
unsigned char *Equalizar_R, *Equalizar_G, *Equalizar_B;
unsigned char *Convol_R, *Convol_G, *Convol_B;
unsigned char *Fourier_R, *Fourier_G, *Fourier_B;
unsigned char *Morfo_R, *Morfo_G, *Morfo_B;
float *Val_Real, *Val_Real_out, *Val_Imag, *Val_Imag_out;
unsigned int *d_his_r;
unsigned int *d_his_g;
unsigned int *d_his_b;
float *DMask;
float *Mask = (float*)malloc(625*sizeof(float));

///////////////////////////////////////////////////
//          Serial part of cuda funtions         //
///////////////////////////////////////////////////

int Threshold(unsigned char *r_data, unsigned char *g_data, unsigned char *b_data, size_t pitch);

void Equalization_PC (unsigned char *r_data, unsigned char *g_data,
                     unsigned char *b_data, size_t pitch,
                     unsigned char *r_dataE, unsigned char *g_dataE,
                     unsigned char *b_dataE );

void FFT();

///////////////////////////////////////////////////
//         Function to display with glut         //
///////////////////////////////////////////////////

void display(){
  GLuint bufferObj;
  struct hipGraphicsResource* resource;
  bool Flag_conv = 1;
  glClearColor( 255.0, 255.0, 255.0, 1.0  );
  glClear( GL_COLOR_BUFFER_BIT );

  glGenBuffers( 1, &bufferObj );
  glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj );
  glBufferData( GL_PIXEL_UNPACK_BUFFER_ARB, widht * height * 4, NULL, GL_DYNAMIC_DRAW_ARB );
  hipGraphicsGLRegisterBuffer( &resource, bufferObj, cudaGraphicsMapFlagsNone );
  uchar4* devPtr;
  size_t size;
  hipGraphicsMapResources( 1, &resource, NULL ) ;
  hipGraphicsResourceGetMappedPointer( (void**)&devPtr, &size, resource );

  dim3 grids(Num_Cols/16,Num_Rows/16);
  dim3 threads(16, 16);
  dim3 grids_01(DIM/16,DIM/16);
  dim3 threads_01(16, 16);
  dim3 grids_02(widht/16,height/16);
  dim3 threads_02(16, 16);
///////////////////////////////////////////////////
//       Cuda functions called by the menu       //
///////////////////////////////////////////////////
  if(Flag_Pix){
    Pixelado<<<grids,threads>>>(Image_R, Image_G, Image_B,or_size, Image_R, Image_G, Image_B);
    Flag_Pix = 0;}
  if(Flag_req){
    Requant<<<grids,threads>>>(Image_R, Image_G, Image_B,or_size, Image_R, Image_G, Image_B);
    Flag_req = 0;}

  if(Flag_Reset){
    Backup<<<grids,threads>>>(Image_R_bk, Image_G_bk, Image_B_bk, or_size,
      Image_R, Image_G, Image_B);
    Flag_Reset = 0;}

  if(Flag_Ero){
    erode<<<grids,threads>>>(Image_R, Image_G, Image_B,
      or_size, Image_R, Image_G, Image_B, Num_Cols, Num_Rows, Dim_Con, Max_E);
    Flag_Ero = 0;}
  if(Flag_Dil){
    dilate<<<grids,threads>>>(Image_R, Image_G, Image_B,
      or_size, Image_R, Image_G, Image_B, Num_Cols, Num_Rows, Dim_Con, Max_E);
    Flag_Dil = 0;}
  if(Flag_Gray){
    grayscale<<<grids,threads>>>(Image_R, Image_G, Image_B, or_size, d_his_r);
    Flag_Gray = 0;}

  if(Flag_BW){
    Threshold (Image_R, Image_G, Image_B, or_size );
    Flag_BW = 0;}

  if (Flag_PPnoise){
    PPnoise<<<grids,threads>>>(Image_R, Image_G, Image_B, or_size, 1, rand()%100);
    Flag_PPnoise = 0;}

  if(Flag_Fourier){
    FFT();
    Flag_Fourier = 0;}

  if (Equalization){
    Equalization_PC (Image_R, Image_G, Image_B, or_size,Convol_R, Convol_G, Convol_B );
    Backup<<<grids,threads>>>(Convol_R, Convol_G, Convol_B, or_size,
      Image_R, Image_G, Image_B); 
    Equalization=0;   
  }
  if (Flag_Med) {
    median_filter<<<grids,threads>>>(Image_R, Image_G, Image_B, or_size,
      Convol_R, Convol_G, Convol_B, Num_Cols, Num_Rows, 3);
    Backup<<<grids,threads>>>(Convol_R, Convol_G, Convol_B, or_size,
      Image_R, Image_G, Image_B);
    Flag_Med=0;
  }
  if (Flag_Filt) {
    Operador_Convolucion<<<grids,threads>>>(Image_R, Image_G, Image_B,
      or_size, Convol_R, Convol_G, Convol_B, Num_Cols, Num_Rows, DMask, Dim_Con);
    Backup<<<grids,threads>>>(Convol_R, Convol_G, Convol_B, or_size,
      Image_R, Image_G, Image_B);
    Flag_Filt=0;
  }

  if (Flag_conv) {
    Scale<<<grids_01,threads_01>>>(Image_R, Image_G, Image_B, Morfo_R, Morfo_G, Morfo_B,
      or_size, mor_size, Scale_Factor, Num_Cols, Num_Rows);
  }else{
    Scale<<<grids_01,threads_01>>>(Convol_R, Convol_G, Convol_B, Morfo_R, Morfo_G, Morfo_B,
      or_size, mor_size, Scale_Factor, Num_Cols, Num_Rows);
  }


  Rotate<<<grids_02,threads_02>>>( devPtr, Morfo_R, Morfo_G, Morfo_B,
                              mor_size, Rotation_Factor, DIM, DIM);

  hipGraphicsUnmapResources( 1, &resource, NULL ) ;
  glDrawPixels( widht, height, GL_RGBA, GL_UNSIGNED_BYTE, 0 );
  glutSwapBuffers();
  hipGraphicsUnregisterResource( resource ) ;
  glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, 0 );
  glDeleteBuffers( 1, &bufferObj );
}

///////////////////////////////////////////////////
//                Serial funtions                //
///////////////////////////////////////////////////
int Threshold(unsigned char *r_data, unsigned char *g_data, unsigned char *b_data, size_t pitch) {
  unsigned int his_size = sizeof(unsigned int)*256;
  unsigned int *his = (unsigned int*)malloc(his_size);

  hipMemset( d_his_r, 0, his_size);
  dim3 grids(Num_Cols,Num_Rows);
  dim3 threads(1, 1);
  grayscale<<<grids,threads>>>(r_data, g_data, b_data, pitch, d_his_r);
  hipMemcpy(his, d_his_r, his_size, hipMemcpyDeviceToHost);
  int m = Num_Cols*Num_Rows/2, h = 0, um, i;
  for (i = 0; i < 256; i++) {
    h += his[i];
    if (h > m) {
      um = i;
      break;
    }
  }
  binary<<<grids,threads>>>(r_data, g_data, b_data, pitch, um);
  return um;
}

void FFT(){
  FFT_X<<<Num_Cols_Fourier/128, 128>>>(Image_R, Image_G, Image_B,
                       or_size, Val_Real, Val_Imag, Val_Real_out, Val_Imag_out,
                       Image_R, Image_G, Image_B, Num_Cols, Num_Rows, Num_Cols_Fourier, Num_Rows_Fourier);

  FFT_Y<<<Num_Rows_Fourier/128, 128>>>(Image_R, Image_G, Image_B,
                       or_size, Val_Real, Val_Imag, Val_Real_out, Val_Imag_out,
                       Image_R, Image_G, Image_B, Num_Cols, Num_Rows, Num_Cols_Fourier, Num_Rows_Fourier);
}

void Equalization_PC (unsigned char *r_data, unsigned char *g_data,
                     unsigned char *b_data, size_t pitch,
                     unsigned char *r_dataE, unsigned char *g_dataE,
                     unsigned char *b_dataE ){
  int i;
  unsigned int his_size = sizeof(unsigned int)*256;
  float hisAc_size = sizeof(float)*256;

  unsigned int *his_r = (unsigned int*)malloc(his_size);
  unsigned int *his_g = (unsigned int*)malloc(his_size);
  unsigned int *his_b = (unsigned int*)malloc(his_size);

  float *hisAc_r = (float*)malloc(hisAc_size);
  float *hisAc_g = (float*)malloc(hisAc_size);
  float *hisAc_b = (float*)malloc(hisAc_size);

  hipMemset( d_his_r, 0, his_size);
  hipMemset( d_his_g, 0, his_size);
  hipMemset( d_his_b, 0, his_size);

  dim3 grids(Num_Cols,Num_Rows);
  dim3 threads(1, 1);
  Get_Histogram<<<grids,threads>>>(r_data, g_data, b_data, pitch, d_his_r, d_his_g, d_his_b);

  hipMemcpy(his_r, d_his_r, his_size, hipMemcpyDeviceToHost);
  hipMemcpy(his_g, d_his_g, his_size, hipMemcpyDeviceToHost);
  hipMemcpy(his_b, d_his_b, his_size, hipMemcpyDeviceToHost);
  float szImage=Num_Cols*Num_Rows;
  hisAc_r[0] = ((float)his_r[0]);
  hisAc_g[0] = ((float)his_g[0]);
  hisAc_b[0] = ((float)his_b[0]);
  for (i = 1; i < 256; i++) {
    hisAc_r[i] = hisAc_r[i-1] + (((float)his_r[i]));
    hisAc_g[i] = hisAc_g[i-1] + (((float)his_g[i]));
    hisAc_b[i] = hisAc_b[i-1] + (((float)his_b[i]));
  }
  his_r[0] = 0;
  his_g[0] = 0;
  his_b[0] = 0;

  for (i = 1; i < 255; i++) {
    his_r[i] = (int)(hisAc_r[i - 1]*255/szImage);
    his_g[i] = (int)(hisAc_g[i - 1]*255/szImage);
    his_b[i] = (int)(hisAc_b[i - 1]*255/szImage);
  }
  his_r[255] = 255;
  his_g[255] = 255;
  his_b[255] = 255;

  hipMemcpy(d_his_r, his_r, his_size, hipMemcpyHostToDevice);
  hipMemcpy(d_his_g, his_g, his_size, hipMemcpyHostToDevice);
  hipMemcpy(d_his_b, his_b, his_size, hipMemcpyHostToDevice);

  Equalization_GPU<<<grids,threads>>>(r_data, g_data, b_data,
    or_size, r_dataE, g_dataE, b_dataE, d_his_r, d_his_g, d_his_b);
}

///////////////////////////////////////////////////
//                 Menu options                  //
///////////////////////////////////////////////////

void call_back_function(int val){
  switch (val) {
    case 2:
      if(Scale_Factor < 1)Scale_Factor = 1/((1/Scale_Factor) + 0.15);
      else Scale_Factor -= 0.15;
      break;
    case 1:
      if(Scale_Factor < 1)Scale_Factor = 1/((1/Scale_Factor) - 0.15);
      else Scale_Factor += 0.15;
      break;
    case 3:
      Rotation_Factor -= 0.01*PI;
      break;
    case 4:
      Rotation_Factor += 0.01*PI;
      break;
    case 18:
      Equalization = 1;
      break;
    case 6:
      Flag_Filt = 1;
      Dim_Con = 3;
      Mask[0] = 1/9.0; Mask[1] = 1/9.0; Mask[2] = 1/9.0;
      Mask[3] = 1/9.0; Mask[4] = 1/9.0; Mask[5] = 1/9.0;
      Mask[6] = 1/9.0; Mask[7] = 1/9.0; Mask[8] = 1/9.0;
      hipMemcpy(DMask, Mask, 9*sizeof(float), hipMemcpyHostToDevice);
      break;
    case 8:
      Flag_Filt = 1;
      Dim_Con = 3;
      Mask[0] = 1/16.0; Mask[1] = 2/16.0; Mask[2] = 1/16.0;
      Mask[3] = 2/16.0; Mask[4] = 4/16.0; Mask[5] = 2/16.0;
      Mask[6] = 1/16.0; Mask[7] = 2/16.0; Mask[8] = 1/16.0;
      hipMemcpy(DMask, Mask, 9*sizeof(float), hipMemcpyHostToDevice);
      break;
    case 9:
      Flag_Filt = 1;
      Dim_Con = 3;
      Mask[0] = 0; Mask[1] = -1; Mask[2] = 0;
      Mask[3] = -1; Mask[4] = 4; Mask[5] = -1;
      Mask[6] = 0; Mask[7] = -1; Mask[8] = 0;
      hipMemcpy(DMask, Mask, 9*sizeof(float), hipMemcpyHostToDevice);
      break;
    case 10:
      Flag_Filt = 1;
      Dim_Con = 3;
      Mask[0] = -1; Mask[1] = -1; Mask[2] = -1;
      Mask[3] = -1; Mask[4] = 8; Mask[5] = -1;
      Mask[6] = -1; Mask[7] = -1; Mask[8] = -1;
      hipMemcpy(DMask, Mask, 9*sizeof(float), hipMemcpyHostToDevice);
      break;
    case 11:
      Flag_Filt = 1;
      Dim_Con = 3;
      Mask[0] = -1; Mask[1] = 0; Mask[2] = 1;
      Mask[3] = -1; Mask[4] = 0; Mask[5] = 1;
      Mask[6] = -1; Mask[7] = 0; Mask[8] = 1;
      hipMemcpy(DMask, Mask, 9*sizeof(float), hipMemcpyHostToDevice);
    case 12:
      Flag_Filt = 1;
      Dim_Con = 3;
      Mask[0] = -1; Mask[1] = 0; Mask[2] = 1;
      Mask[3] = -2; Mask[4] = 0; Mask[5] = 2;
      Mask[6] = -1; Mask[7] = 0; Mask[8] = 1;
      hipMemcpy(DMask, Mask, 9*sizeof(float), hipMemcpyHostToDevice);
      break;
    case 13:
      Flag_Filt = 1;
      Dim_Con = 3;
      Mask[0] = 1; Mask[1] = 2; Mask[2] = 1;
      Mask[3] = 0; Mask[4] = 0; Mask[5] = 0;
      Mask[6] = -1; Mask[7] = -2; Mask[8] = -1;
      hipMemcpy(DMask, Mask, 9*sizeof(float), hipMemcpyHostToDevice);
    case 5:
      Flag_PPnoise = 1;
      break;
    case 7:
      Flag_Med = 1;
      break;
    case 19:
      Flag_Fourier = 1;
      break;
    case 20:
      Flag_Reset = 1;
      Scale_Factor = 1;
      Rotation_Factor = 0;
      break;
    case 16:
      Flag_Ero = 1;
      Dim_Con = 3;
      Max_E = 255;
      break;
    case 17:
      Flag_Dil = 1;
      Dim_Con = 3;
      break;
    case 14:
      Flag_Gray = 1;
      break;
    case 15:
      Flag_BW = 1;
      break;
    case 21:
      exit(0);
      break;
    case 22:
      Flag_Pix = 1;
      break;
    case 23:
      Flag_req = 1;
      break;
    default:{
      }
    }
  display();
}

///////////////////////////////////////////////////
//                 Creating menu                 //
///////////////////////////////////////////////////
void Create_call_back_function(void) {
	sub_00 = glutCreateMenu(call_back_function);
	glutAddMenuEntry("Acercar", 1);
	glutAddMenuEntry("Alejar", 2);
        glutAddMenuEntry("Rotar derecha", 3);
        glutAddMenuEntry("Rotar izquierda", 4);

	sub_01 = glutCreateMenu(call_back_function);
	glutAddMenuEntry("Agregar ruido", 5);
	glutAddMenuEntry("Filtro de media", 6);
        glutAddMenuEntry("Filtro de mediana", 7);
        glutAddMenuEntry("Filtro gaussiano", 8);

	sub_02 = glutCreateMenu(call_back_function);
	glutAddMenuEntry("2D - 4 conexion", 9);
	glutAddMenuEntry("2D - 8 conexion", 10);
        glutAddMenuEntry("Prewitt", 11);
        glutAddMenuEntry("Sobel X", 12);
        glutAddMenuEntry("Sobel Y", 13);

        sub_03 = glutCreateMenu(call_back_function);
	glutAddMenuEntry("Escala de grises", 14);
	glutAddMenuEntry("Binarizado", 15);

        sub_04 = glutCreateMenu(call_back_function);
        glutAddMenuEntry("Erosion", 16);
        glutAddMenuEntry("Dilatacion", 17);

        glutCreateMenu(call_back_function);
	glutAddMenuEntry("Pixelado", 22);
        glutAddMenuEntry("Recuantizacion", 23);
        glutAddSubMenu("Rotacion-escala", sub_00);
        glutAddSubMenu("Ruido-suavizado", sub_01);
	glutAddSubMenu("Deteccion de bordes", sub_02);
	glutAddSubMenu("Sistemas de color", sub_03);
        glutAddSubMenu("Operaciones morfologicas", sub_04);
        glutAddMenuEntry("Ecualizacion", 18);
        glutAddMenuEntry("Transformacion Fourier", 19);
        glutAddMenuEntry("Restaurar original", 20);
        glutAddMenuEntry("Salir", 21);
	glutAttachMenu(GLUT_RIGHT_BUTTON);
}


///////////////////////////////////////////////////
//  Struct and function to read data from image  //
///////////////////////////////////////////////////

typedef struct BMP_Info{
  unsigned long bytesInHeader;
  unsigned long widht;
  unsigned long height;
  unsigned int planes;
  unsigned int bitsPerPixel;
  unsigned long compression;
  unsigned long sizeImage;
  unsigned long hResolution;
  unsigned long vResolution;
  unsigned long nIndexes;
  unsigned long nIIndexes;
  char type[3];
  unsigned long size;
  char reserved[5];
  unsigned long offset;
} BMP_Info;

unsigned long Turn_Data_Long(FILE* fp){
  uint32_t data32;
  fread (&(data32),4, 1,fp);
  unsigned long data = (unsigned long)data32;
  return data;
}

unsigned int Turn_Data_Int(FILE* fp){
  uint16_t data16;
  fread (&(data16), 2, 1, fp);
  unsigned int data = (unsigned int)data16;
  return data;
}

void Read_Image(FILE* fp, BMP_Info* Image_Raw){
  fgets(Image_Raw->type, 3, fp);
  Image_Raw->size = Turn_Data_Long(fp);
  fgets(Image_Raw->reserved, 5, fp);
  Image_Raw->offset = Turn_Data_Long(fp);
  Image_Raw->bytesInHeader = Turn_Data_Long(fp);
  Image_Raw->widht = Turn_Data_Long(fp);
  Image_Raw->height = Turn_Data_Long(fp);
  Image_Raw->planes = Turn_Data_Int(fp);
  Image_Raw->bitsPerPixel = Turn_Data_Int(fp);
  Image_Raw->compression = Turn_Data_Long(fp);
  Image_Raw->sizeImage = Turn_Data_Long(fp);
  Image_Raw->hResolution = Turn_Data_Long(fp);
  Image_Raw->vResolution = Turn_Data_Long(fp);
  Image_Raw->nIndexes = Turn_Data_Long(fp);
  Image_Raw->nIIndexes = Turn_Data_Long(fp);
}

FILE *fp;
BMP_Info Image_Raw;
