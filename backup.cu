
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define DIM 1600
#define PI 3.14159265

__global__ void Backup(unsigned char *R_input, unsigned char *G_input,
                      unsigned char *B_input, size_t i_size,
                      unsigned char *R_output, unsigned char *G_output,
                      unsigned char *B_output){
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int offset = x + y * i_size;
  R_output[offset] = R_input[offset];
  G_output[offset] = G_input[offset];
  B_output[offset] = B_input[offset];
}



