
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define DIM 1600
#define PI 3.14159265

__device__ int log2(int N){
  int k = N, i = 0;
  while(k) {
    k >>= 1;
    i++;}
  return i - 1;
}

__device__ int reverse(int N, int n) {
  int p = 0;
  for(int j = 1; j <= log2(N); j++) {
    if(n & (1 << (log2(N) - j)))
      p |= 1 << (j - 1);
  }
  return p;
}


__device__ void ordina_x(float *complex_r, float *complex_i,
                      float *real_d_out, float *imagi_d_out,
                      int row, int col, int x) {
  int N = row, a;
  for(int i = 0; i < N; i++){
    a = reverse((int)N, i);
    real_d_out[i*col + x] = complex_r[a*col + x];
    imagi_d_out[i*col + x] = complex_i[a*col + x];}
  for(int j = 0; j < N; j++){
    complex_r[j*col + x] = real_d_out[j*col + x];
    complex_i[j*col + x] = imagi_d_out[j*col + x];}
}

__device__ void ordina_y(float *complex_r, float *complex_i,
                      float *real_d_out, float *imagi_d_out,
                      int row, int col, int y) {
  int N = row, a;
  for(int i = 0; i < N; i++){
    a = reverse((int)N, i);
    real_d_out[y*col + i] = complex_r[y*col + a];
    imagi_d_out[y*col + i] = complex_i[y*col + a];}
  for(int j = 0; j < N; j++){
    complex_r[y*col + j] = real_d_out[y*col + j];
    complex_i[y*col + j] = imagi_d_out[y*col + j];}
}

__device__ void Func_FFT_X(float *complex_r, float *complex_i,
                     int row, int col, int x){
  int n = 1, N = row;
  int a = N/2;
  float temp_real, temp_imagi;
  float t_r, t_i, a_r, a_i;
  for(int j = 0; j < log2(N); j++){
    for (int i = 0; i < N; i++) {
      if(!(i & n)) {
        temp_real = complex_r[x + (i * col)];
        temp_imagi = complex_i[x + (i * col)];
        a_r = cos((-2) * ((i * a) % (n * a)) * PI / N);
        a_i = sin((-2) * ((i * a) % (n * a)) * PI / N);
        t_r = (a_r*complex_r[x + (i + n)*col]) - (a_i*complex_i[x + (i + n)*col]);
        t_i = (a_i*complex_r[x + (i + n)*col]) + (a_r*complex_i[x + (i + n)*col]);
        complex_r[x + (i * col)] += t_r;
        complex_i[x + (i * col)] += t_i;
        complex_r[x + (i + n)*col] = temp_real - t_r;
        complex_i[x + (i + n)*col] = temp_imagi - t_i;}
    }
    n *= 2;
    a = a/2;
  }
}

__device__ void Func_FFT_Y(float *complex_r, float *complex_i,
                     int row, int col, int y){
  int n = 1, N = col;
  int a = N/2;
  float temp_real, temp_imagi;
  float t_r, t_i, a_r, a_i;
  for(int j = 0; j < log2(N); j++){
    for (int i = 0; i < N; i++) {
      if(!(i & n)) {
        temp_real = complex_r[i + (y * col)];
        temp_imagi = complex_i[i + (y * col)];
        a_r = cos(-2 * ((i * a) % (n * a)) * PI/ N);
        a_i = sin(-2 * ((i * a) % (n * a)) * PI/ N);
        t_r = (a_r*complex_r[(i + n) + y*col]) - (a_i*complex_i[(i + n) + y*col]);
        t_i = (a_i*complex_r[(i + n) + y*col]) + (a_r*complex_i[(i + n) + y*col]);
        complex_r[i + (y * col)] += t_r;
        complex_i[i + (y * col)] += t_i;
        complex_r[(i + n) + y*col] = temp_real - t_r;
        complex_i[(i + n) + y*col] = temp_imagi - t_i;}
    }
    n *= 2;
    a = a/2;
  }
}

__global__ void FFT_X(unsigned char *R_input, unsigned char *G_input,
                    unsigned char *B_input, size_t i_size,
                    float *complex_r, float *complex_i,
                    float *real_d_out, float *imagi_d_out,
                    unsigned char *r_dataC, unsigned char *g_dataC,
                    unsigned char *b_dataC, unsigned long col, unsigned long row,
                    unsigned long colF, unsigned long rowF ) {
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  float temp;
  if(x < col){
    for (int i = 0; i < row; i++) {
      complex_r[x + (i * colF)] = 0.2989 * R_input[x + (i * i_size)] +  0.587 * G_input[x + (i * i_size)] + 0.1140 * B_input[x + (i * i_size)];
      complex_i[x + (i * colF)] = 0;}
    for (int i = row; i < rowF; i++) {
      complex_r[x + (i * colF)] = 0;
      complex_i[x + (i * colF)] = 0;}
  }else{
    for (int i = 0; i < rowF; i++) {
      complex_r[x + (i * colF)] = 0;
      complex_i[x + (i * colF)] = 0;}
  }
  ordina_x(complex_r, complex_i, real_d_out, imagi_d_out, rowF, colF, x);
  Func_FFT_X(complex_r, complex_i, rowF, colF, x);
  for (int i = 0; i < rowF/2; i++){
    temp = complex_r[x + (i * colF)];
    complex_r[x + (i * colF)] = complex_r[x + ((i + rowF/2) * colF)];
    complex_r[x + ((i + rowF/2) * colF)] = temp;
    temp = complex_i[x + (i * colF)];
    complex_i[x + (i * colF)] = complex_i[x + ((i + rowF/2) * colF)];
    complex_i[x + ((i + rowF/2) * colF)] = temp;}
}

__global__ void FFT_Y(unsigned char *R_input, unsigned char *G_input,
                    unsigned char *B_input, size_t i_size,
                    float *complex_r, float *complex_i,
                    float *real_d_out, float *imagi_d_out,
                    unsigned char *r_dataC, unsigned char *g_dataC,
                    unsigned char *b_dataC, unsigned long col, unsigned long row,
                    unsigned long colF, unsigned long rowF ) {
  int y = threadIdx.x + (blockIdx.x * blockDim.x);
  float temp;
  ordina_y(complex_r, complex_i, real_d_out, imagi_d_out, rowF, colF, y);
  Func_FFT_Y(complex_r, complex_i, rowF, colF, y);
  for (int i = 0; i < colF/2; i++) {
    temp = complex_r[i + (y * colF)];
    complex_r[i + (y * colF)] = complex_r[(i + colF/2) + (y * colF)];
    complex_r[(i + colF/2) + (y * colF)] = temp;
    temp = complex_i[i + (y * colF)];
    complex_i[i + (y * colF)] = complex_i[(i + colF/2) + (y * colF)];
    complex_i[(i + colF/2) + (y * colF)] = temp;}

  unsigned char v;
  int a = (colF/2) - (col/2);
  int temp_b = (rowF/2) - (row/2);
  if( y >= temp_b)
    for (int i = a; i < (colF/2) + (col/2); i++) {
      v = (unsigned char)(20*log10(sqrt((complex_r[i + (y * colF)]*complex_r[i + (y * colF)]) + (complex_i[i + (y * colF)]*complex_i[i + (y * colF)]))));
      r_dataC[(i - a ) + (y - temp_b) * i_size] = v;
      g_dataC[(i - a) + (y - temp_b) * i_size] = v;
      b_dataC[(i - a) + (y - temp_b) * i_size] = v;}
}



