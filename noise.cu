
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define DIM 1600
#define PI 3.14159265

__global__ void PPnoise(unsigned char *R_input, unsigned char *G_input,
                      unsigned char *B_input, size_t i_size, int noiseP, int seed){
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int offset = x + y * i_size;
  hiprandState_t state;
  hiprand_init(seed, x,  y, &state);

  unsigned char noise = (unsigned char)(hiprand(&state) % 100);
  if(hiprand(&state) % 100 < noiseP){
    noise = 255 * (noise % 2);
    R_input[offset] = noise;
    G_input[offset] = noise;
    B_input[offset] = noise;
  }
}
