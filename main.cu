#include "hip/hip_runtime.h"
///////////////////////////////////////////////////
//          Call to general libraries            //
///////////////////////////////////////////////////
#include <cstdio>
#include <stdlib.h>
#include <stdint.h>
#include <assert.h>
#include <math.h>
#include <complex.h>
#include <string.h>
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <GL/glut.h>
#include <GL/freeglut_ext.h>
#include <cuda_gl_interop.h>
#include "UI.cu"
using namespace std;


///////////////////////////////////////////////////
//                  Main function                //
///////////////////////////////////////////////////
int main(int argc, char** argv){
  time_t t;
  srand((unsigned) time(&t));
  int i = 0, j;
  unsigned char c;
  Scale_Factor = 1;
  Rotation_Factor = 0;
  //////// Reading data from images
  char T[1024]="montaña.bmp";
  fp = fopen(T,"r");
  Read_Image(fp, &Image_Raw);
  i = 54;
  while(i < Image_Raw.offset) {
    c = fgetc(fp);
    if(feof(fp))
       break;
    i++;
  }
  ////////Building RGB matrix 
  Num_Cols = Image_Raw.widht, Num_Rows = Image_Raw.height;
  sizeImage = Num_Cols*Num_Rows;
  unsigned char RR[Num_Rows][Num_Cols], GG[Num_Rows][Num_Cols], BB[Num_Rows][Num_Cols];
  for ( i = 0; i < Num_Rows; i++) {
    for( j = 0; j < Num_Cols; j++){
      if (Image_Raw.bitsPerPixel > 8) {
        BB[i][j] = fgetc(fp);
        GG[i][j] = fgetc(fp);
        RR[i][j] = fgetc(fp);
        if(Image_Raw.bitsPerPixel > 24)
          c = fgetc(fp);
      } else if(Image_Raw.bitsPerPixel == 8) {
        c = getc(fp);
        BB[i][j] = c;
        GG[i][j] = c;
        RR[i][j] = c;}
    }
  }
  fclose(fp);

  widht = WIDTH; height = HEIGHT;

  Num_Rows_Fourier = pow(2,(int)(log(Num_Rows - 1)/log(2)) + 1);
  Num_Cols_Fourier = pow(2,(int)(log(Num_Cols - 1)/log(2)) + 1);
  unsigned int his_size = sizeof(unsigned int)*256;
  unsigned int comp_size = sizeof(float)*Num_Rows_Fourier*Num_Cols_Fourier;

  hipMallocManaged(&d_his_r, his_size);
  hipMallocManaged(&d_his_g, his_size);
  hipMallocManaged(&d_his_b, his_size);

  hipMallocManaged(&Val_Real, comp_size);
  hipMallocManaged(&Val_Real_out, comp_size);
  hipMallocManaged(&Val_Imag, comp_size);
  hipMallocManaged(&Val_Imag_out, comp_size);

  hipMallocManaged(&DMask, sizeof(float)*625);
///////////////////////////////////////////////////
//                  Cuda memory                 //
///////////////////////////////////////////////////
  // Original
  hipMallocPitch((void**)&Image_R, &or_size, sizeof(unsigned char)*Num_Cols, Num_Rows);
  hipMallocPitch((void**)&Image_G, &or_size, sizeof(unsigned char)*Num_Cols, Num_Rows);
  hipMallocPitch((void**)&Image_B, &or_size, sizeof(unsigned char)*Num_Cols, Num_Rows);
  // Back - up
  hipMallocPitch((void**)&Image_R_bk, &or_size, sizeof(unsigned char)*Num_Cols, Num_Rows);
  hipMallocPitch((void**)&Image_G_bk, &or_size, sizeof(unsigned char)*Num_Cols, Num_Rows);
  hipMallocPitch((void**)&Image_B_bk, &or_size, sizeof(unsigned char)*Num_Cols, Num_Rows);
  // Fourier
  hipMallocPitch((void**)&Fourier_R, &or_size, sizeof(unsigned char)*Num_Cols, Num_Rows);
  hipMallocPitch((void**)&Fourier_G, &or_size, sizeof(unsigned char)*Num_Cols, Num_Rows);
  hipMallocPitch((void**)&Fourier_B, &or_size, sizeof(unsigned char)*Num_Cols, Num_Rows);
  hipMallocPitch((void**)&Val_Real, &fou_size, sizeof(float)*Num_Cols, Num_Rows);
  hipMallocPitch((void**)&Val_Imag, &fou_size, sizeof(float)*Num_Cols, Num_Rows);
  // Convolution
  hipMallocPitch((void**)&Convol_R, &or_size, sizeof(unsigned char)*Num_Cols, Num_Rows);
  hipMallocPitch((void**)&Convol_G, &or_size, sizeof(unsigned char)*Num_Cols, Num_Rows);
  hipMallocPitch((void**)&Convol_B, &or_size, sizeof(unsigned char)*Num_Cols, Num_Rows);
  // Equalization
  hipMallocPitch((void**)&Equalizar_R, &equ_size, sizeof(unsigned char)*Num_Cols, Num_Rows);
  hipMallocPitch((void**)&Equalizar_G, &equ_size, sizeof(unsigned char)*Num_Cols, Num_Rows);
  hipMallocPitch((void**)&Equalizar_B, &equ_size, sizeof(unsigned char)*Num_Cols, Num_Rows);
  // Rotate and scaling
  hipMallocPitch((void**)&Morfo_R, &mor_size, sizeof(unsigned char)*DIM, DIM);
  hipMallocPitch((void**)&Morfo_G, &mor_size, sizeof(unsigned char)*DIM, DIM);
  hipMallocPitch((void**)&Morfo_B, &mor_size, sizeof(unsigned char)*DIM, DIM);

  // Copying to GPU
  hipMemcpy2D(Image_R, or_size, RR, sizeof(unsigned char)*Num_Cols,
               sizeof(unsigned char)*Num_Cols, Num_Rows, hipMemcpyHostToDevice);

  hipMemcpy2D(Image_G, or_size, GG, sizeof(unsigned char)*Num_Cols,
               sizeof(unsigned char)*Num_Cols, Num_Rows, hipMemcpyHostToDevice);

  hipMemcpy2D(Image_B, or_size, BB, sizeof(unsigned char)*Num_Cols,
               sizeof(unsigned char)*Num_Cols, Num_Rows, hipMemcpyHostToDevice);

  hipMemcpy2D(Image_R_bk, or_size, RR, sizeof(unsigned char)*Num_Cols,
               sizeof(unsigned char)*Num_Cols, Num_Rows, hipMemcpyHostToDevice);

  hipMemcpy2D(Image_G_bk, or_size, GG, sizeof(unsigned char)*Num_Cols,
               sizeof(unsigned char)*Num_Cols, Num_Rows, hipMemcpyHostToDevice);

  hipMemcpy2D(Image_B_bk, or_size, BB, sizeof(unsigned char)*Num_Cols,
               sizeof(unsigned char)*Num_Cols, Num_Rows, hipMemcpyHostToDevice);

  glutInitWindowSize(widht, height);
  glutInit(&argc, argv);
  glutInitContextFlags(GLUT_DEBUG);
  glutInitDisplayMode(GLUT_RGBA | GLUT_DEPTH | GLUT_STENCIL | GLUT_DOUBLE);
  glutCreateWindow("Editor de Imagenes");

  glewInit();

  if (GLEW_KHR_debug){
    glEnable(GL_DEBUG_OUTPUT);
    glEnable(GL_DEBUG_OUTPUT_SYNCHRONOUS);
  } else {
    printf("No GLEW_KHR_debug!");}

  Create_call_back_function();
  glutDisplayFunc(display);
  glutMainLoop();

  hipFree(Image_R), hipFree(Image_G), hipFree(Image_B);
  hipFree(Morfo_R), hipFree(Morfo_G), hipFree(Morfo_B);
  hipFree(Fourier_R), hipFree(Fourier_G), hipFree(Fourier_B);
  hipFree(Convol_R), hipFree(Convol_G), hipFree(Convol_B);
  hipFree(Val_Real), hipFree(Val_Real);
  return 0;

}
